#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <conio.h>

//CPU - host
//GPU - device

//blockDim  - size of block
//blockIdx  - index of current block
//threadIdx - index of current thread in block

__device__ void elem(double* ar, int m, int n, double k, int N) //execution on Device
{
    int tid = blockIdx.y * blockDim.y + threadIdx.y;
    if (tid < N) 
        ar[m * N + tid] -= k * ar[n * N + tid];
}

__global__ void triangle_kernel(double* arr, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int j;
    double kof;
    for (j = 0; j < N * N; j++)
    {
        // if (!arr[j * N + j]) elem(arr, j, N - 1, N, N);
        if (tid >= j && tid < N - 1)
        {
            kof = arr[(tid + 1) * N + j] / arr[j * N + j];
            elem(arr, tid + 1, j, kof, N);
        }
    }
}


//����������� �������� �� ������� ��������� ��� ����������� �������, ��� ����� �������� ������������
double det(double* arr, int N)
{
    double d = 1.0;
    for (int i = 0; i < N; i++)  d *= arr[i * N + i];
    return d;
}


void print_cuda_device_info(hipDeviceProp_t& prop)
{
    printf("Device name:                                        %s\n", prop.name);
    printf("Global memory available on device:                  %zu\n", prop.totalGlobalMem);
    printf("Shared memory available per block:                  %zu\n", prop.sharedMemPerBlock);
    printf("Warp size in threads:                               %d\n", prop.warpSize);
    printf("Maximum number of threads per block:                %d\n", prop.maxThreadsPerBlock);
    printf("Maximum size of each dimension of a block[0]:       %d\n", prop.maxThreadsDim[0]);
    printf("Maximum size of each dimension of a block[1]:       %d\n", prop.maxThreadsDim[1]);
    printf("Maximum size of each dimension of a block[2]:       %i\n", prop.maxThreadsDim[2]);
    printf("Maximum size of each dimension of a grid[0]:        %i\n", prop.maxGridSize[0]);
    printf("Maximum size of each dimension of a grid[1]:        %i\n", prop.maxGridSize[1]);
    printf("Maximum size of each dimension of a grid[2]:        %i\n", prop.maxGridSize[2]);
    printf("Clock frequency in kilohertz:                       %i\n", prop.clockRate);
    printf("totalConstMem:                                      %zu\n", prop.totalConstMem);
    printf("Major compute capability:                           %i\n", prop.major);
    printf("Minor compute capability:                           %i\n", prop.minor);
    printf("Number of multiprocessors on device:                %i\n", prop.multiProcessorCount);
}

__host__ int main()
{
    int N;
    printf("Input size of matrix N = ");
    scanf_s("%i", &N);
    unsigned int timer;


    int Matrix_size = N * N;//Size of matrix
    int MatrixTotalMemory = Matrix_size * sizeof(double);//������, ����������� ��� ������� �� GPU 
    double* InputMatrix = new double[Matrix_size];//�������� ������ ��� ������

    //��������� ������� ���������� ������� � ������� �� �����
    srand(time(NULL));
    for (int i = 0; i < Matrix_size; i++)
    {
        InputMatrix[i]  = 1 + rand() % 9;
    }

    printf("\n");
    for (int i = 0; i < Matrix_size; i++)
    {
        printf("%0.2f ", InputMatrix[i]);
        if (((i + 1) % N == 0) && (i != 0)) printf("\n");
    }
    printf("\n");
    _getch();


    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    //print_cuda_device_info(prop);
    double* MatrixDeviceMemory;
    hipMalloc((void**)&MatrixDeviceMemory, MatrixTotalMemory);//�������� ������ ��� ������ �� GPU
    hipMemcpy(MatrixDeviceMemory, InputMatrix, MatrixTotalMemory, hipMemcpyHostToDevice);//�������� �������� ������� �� GPU 
    dim3 gridSize = dim3(N, N, 1);//����������� ����� ������ (dim3), ���������� ��� ��������
    dim3 blockSize = dim3(1, 1, 1);//������ ����� (dim3), ����������� ��� ��������


    //�������������� ���������� ��� ������ ������� ������
    float recording;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //float start2 = clock(); //Fix the begin of work in timeline.

    triangle_kernel <<< gridSize, blockSize >>> (MatrixDeviceMemory, N); //Execution of matrix triangling
    hipDeviceSynchronize();
    hipEventSynchronize(stop); 

    //float end = clock();   //Fix the end of execution

    //�������� ����� ������
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&recording, start, stop);

    hipMemcpy(InputMatrix, MatrixDeviceMemory, MatrixTotalMemory, hipMemcpyDeviceToHost);//�������� ����� ������� � GPU �� CPU
    

    //������� ���������� �������
    int string = 0;

    for (int i = 0; i < Matrix_size; i++)
    {
        if (string && i == string * N)
        {
            int m = i;
            for (int j = string * N; j < string * N + string; j++)
            {
                printf("0.00 ");
                m++;
            }
            i = m;
        }

        printf("%.2f ", InputMatrix[i]);

        if ((i + 1) % N == 0)
        {
            printf("\n");
            string++;
        }
    }


    printf("\ndet A = %.2f \n", det(InputMatrix, N));
    //if (recording > 0)
        printf("Time of execution =  %.2f\n", recording);
    //else printf("Time of execution =  %.2f\n", end - start2);

    hipFree(MatrixDeviceMemory); //Make the memory free
    return 0;
}
