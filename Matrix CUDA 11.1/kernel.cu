#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <conio.h>

//CPU - host
//GPU - device

//blockDim  - dimention of block
//blockIdx  - index of current block
//threadIdx - index of current thread in block

__device__ void elem(double* A, int m, int n, double kof, int N) //execution on Device
{
    int tid = blockIdx.y * blockDim.y + threadIdx.y;
    if (tid < N) 
        A[m * N + tid] -= kof * A[n * N + tid];
}

__global__ void triangle_kernel(double* A, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int j;
    double kof;
    for (j = 0; j < N * N; j++)
    {
        if (tid < N - 1 && tid >= j)
        {
            kof = A[(tid + 1) * N + j] / A[j * N + j];
            elem(A, tid + 1, j, kof, N);
        }
    }
}


double det(double* arr, int N)
{
    double d = 1.0;
    for (int i = 0; i < N; i++)  d *= arr[i * N + i];
    return d;
}


void print_cuda_device_info(hipDeviceProp_t& prop)
{
    printf("Device... ... ...initialized!");
    printf("Device name:                                        %s\n", prop.name);
    //printf("Global memory available on device:                  %zu\n", prop.totalGlobalMem);
    //printf("Shared memory available per block:                  %zu\n", prop.sharedMemPerBlock);
    printf("Warp size in threads:                               %d\n", prop.warpSize);
    printf("Maximum number of threads per block:                %d\n", prop.maxThreadsPerBlock);
    /*
    printf("Maximum size of each dimension of a block[0]:       %d\n", prop.maxThreadsDim[0]);
    printf("Maximum size of each dimension of a block[1]:       %d\n", prop.maxThreadsDim[1]);
    printf("Maximum size of each dimension of a block[2]:       %i\n", prop.maxThreadsDim[2]);
    */
    printf("Maximum size of each dimension of a grid[0]:        %i\n", prop.maxGridSize[0]);
    /*
    printf("Maximum size of each dimension of a grid[1]:        %i\n", prop.maxGridSize[1]);
    printf("Maximum size of each dimension of a grid[2]:        %i\n", prop.maxGridSize[2]);
    printf("Clock frequency in kilohertz:                       %i\n", prop.clockRate);
    printf("totalConstMem:                                      %zu\n", prop.totalConstMem);
    printf("Major compute capability:                           %i\n", prop.major);
    printf("Minor compute capability:                           %i\n", prop.minor);
    */
    printf("Number of multiprocessors on device:                %i\n", prop.multiProcessorCount);
}

__host__ int main()
{
    int N;
    printf("Input size of matrix N = ");
    scanf_s("%i", &N);
    unsigned int timer;


    int Matrix_size = N * N; //Size of matrix
    int MatrixTotalMemory = Matrix_size * sizeof(double);//������, ����������� ��� ������� �� GPU 
    double* InputMatrix = new double[Matrix_size];//�������� ������ ��� ������

    //��������� ������� ���������� ������� � ������� �� �����
    srand(time(NULL));
    for (int i = 0; i < Matrix_size; i++)
    {
        InputMatrix[i]  = 1 + rand() % 9;
    }

    printf("\n");
    for (int i = 0; i < Matrix_size; i++)
    {
        printf("(%0.0f)", InputMatrix[i]);
        if (((i+1) % N == 0) && (i != 0)) 
            printf("\n");
    }
    printf("\n");
    _getch();


    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    //print_cuda_device_info(prop);
    double* MatrixDeviceMemory;
     
    dim3 gridSize = dim3(N, N, 1);  //Dimention of Grid (matrix N*N)
    dim3 blockSize = dim3(1, 1, 1); //Dimention of block 


    //�������������� ���������� ��� ������ ������� ������
    float run_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void**)&MatrixDeviceMemory, MatrixTotalMemory); //allocating memory on GPU
    hipMemcpy(MatrixDeviceMemory, InputMatrix, MatrixTotalMemory, hipMemcpyHostToDevice); //copying operands to GPU

    //float start2 = clock(); //Fix the begin of work in timeline.

    triangle_kernel <<< gridSize, blockSize >>> (MatrixDeviceMemory, N); //Execution of matrix triangling
    hipDeviceSynchronize();
    hipEventSynchronize(stop); 

    //float end = clock();   //Fix the end of execution

    hipMemcpy(InputMatrix, MatrixDeviceMemory, MatrixTotalMemory, hipMemcpyDeviceToHost);//�������� ����� ������� � GPU �� CPU

    //�������� ����� ������
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&run_time, start, stop);

    
    

    //������� ���������� �������
    int string = 0;

    for (int i = 0; i < Matrix_size; i++)
    {
        if (string && i == string * N)
        {
            int m = i;
            for (int j = string * N; j < string * N + string; j++)
            {
                printf("0.00 ");
                m++;
            }
            i = m;
        }

        printf("%.2f ", InputMatrix[i]);

        if ((i + 1) % N == 0)
        {
            printf("\n");
            string++;
        }
    }


    printf("\ndet A = %.2f \n", det(InputMatrix, N));
    //if (recording > 0)
        printf("Time of execution =  %.2f\n", run_time);
    //else printf("Time of execution =  %.2f\n", end - start2);

    hipFree(MatrixDeviceMemory); //Make the memory free
    return 0;
}
